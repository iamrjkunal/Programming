#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void number_xor(int* d_out, int* d_in,int num){
        int myid = blockDim.x * blockIdx.x + threadIdx.x;
        int tid = threadIdx.x;
        for( int i = blockDim.x/2;i>0;i=i/2){
          if(tid<i && (myid +i)<num){
            d_in[myid] ^= d_in[myid + i];
          }
        __syncthreads();
        }
        if(tid==0){
        atomicXor(d_out,d_in[myid]);
        }

}

int main(int argc, char** argv){
        int size= atoi(argv[1]);
        int seed = atoi(argv[2]);
        srand(seed);
	 int h_in[size];
        for(int i=0; i<size; i++) {
                h_in[i]= (int)rand();
        }
        int h_out=0;
        int* d_in;
        int* d_out;

        hipMalloc ((void**)&d_in, size* sizeof(int));
        hipMalloc ((void**)&d_out, sizeof(int));
        hipMemcpy(d_out,&h_out, sizeof(int),hipMemcpyHostToDevice);
	 hipMemcpy(d_in, h_in, size* sizeof(int), hipMemcpyHostToDevice);
        int block = (size/1024);
        if(size%1024!=0) block ++;
        number_xor<<<block,1024>>>(d_out, d_in,size);
        hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
	 printf("%d\n",h_out);
        hipFree(d_out);
        hipFree(d_in);
        return 0;
}
